
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define FILTER_rows 9
__constant__ float dc_filter[FILTER_rows * FILTER_rows];

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, int &rows, int &columns, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &rows);
	fscanf(f, "%i", &columns);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(rows * columns * sizeof(uchar3));
	for (int i = 0; i < rows * columns; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uchar3 * pixels, int rows, int columns, char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", rows, columns); 

	for (int i = 0; i < rows * columns; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

// __global__ void blurImgKernel1(uchar3 * inPixels, int rows, int columns, 
//         float * filter, int filterrows, 
//         uchar3 * outPixels)
// {
// 	// TODO

// }

// __global__ void blurImgKernel2(uchar3 * inPixels, int rows, int columns, 
//         float * filter, int filterrows, 
//         uchar3 * outPixels)
// {
// 	// TODO

// }

// __global__ void blurImgKernel3(uchar3 * inPixels, int rows, int columns, 
//         int filterrows, 
//         uchar3 * outPixels)
// {
// 	// TODO

// }							
void rgbToGray(uchar3 *pixels, int rows, int columns, unsigned char *&grayPixels) {
    grayPixels = (unsigned char *)malloc(rows * columns * sizeof(unsigned char));

    for (int i = 0; i < rows * columns; i++) {
        // Chuyển đổi thành giá trị gray theo công thức cụ thể
        grayPixels[i] = (unsigned char)(0.299 * pixels[i].x + 0.587 * pixels[i].y + 0.114 * pixels[i].z);
    }
}
void writePnmGray(char *fileName, int rows, int columns, unsigned char *grayPixels) {
    FILE *f = fopen(fileName, "w");
    if (f == NULL) {
        printf("Cannot write %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    fprintf(f, "P2\n");
    fprintf(f, "%d %d\n", rows, columns);
    fprintf(f, "255\n");

    for (int i = 0; i < rows * columns; i++) {
        fprintf(f, "%d\n", grayPixels[i]);
    }

    fclose(f);
}

void convolveX(unsigned char *inputPixels, int rows, int columns, unsigned char *&outputPixels) {
    // Sobel filter for x-direction
    int sobelFilter[3][3] = {{1, 0, -1}, {2, 0, -2}, {1, 0, -1}};

    outputPixels = (unsigned char *)malloc(rows * columns * sizeof(unsigned char));

    // Convolution
    for (int y = 1; y < columns - 1; y++) {
        for (int x = 1; x < rows - 1; x++) {
            int sum = 0;
            for (int i = -1; i <= 1; i++) {
                for (int j = -1; j <= 1; j++) {
                    sum += sobelFilter[i + 1][j + 1] * inputPixels[(y + i) * rows + (x + j)];
                }
            }
            // Ensure the result is within the valid range [0, 255]
            outputPixels[y * rows + x] = (unsigned char)(sum > 255 ? 255 : (sum < 0 ? 0 : sum));
        }
    }
}

void convolveY(unsigned char *inputPixels, int rows, int columns, unsigned char *&outputPixels) {
    // Sobel filter for y-direction
    int sobelFilter[3][3] = {{1, 2, 1}, {0, 0, 0}, {-1, -2, -1}};

    outputPixels = (unsigned char *)malloc(rows * columns * sizeof(unsigned char));

    // Convolution
    for (int y = 1; y < columns - 1; y++) {
        for (int x = 1; x < rows - 1; x++) {
            int sum = 0;
            for (int i = -1; i <= 1; i++) {
                for (int j = -1; j <= 1; j++) {
                    sum += sobelFilter[i + 1][j + 1] * inputPixels[(y + i) * rows + (x + j)];
                }
            }
            // Ensure the result is within the valid range [0, 255]
            outputPixels[y * rows + x] = (unsigned char)(sum > 255 ? 255 : (sum < 0 ? 0 : sum));
        }
    }
}

void calculateImportance(unsigned char *edgesX, unsigned char *edgesY, int rows, int columns, unsigned char *&importance) {
    importance = (unsigned char *)malloc(rows * columns * sizeof(unsigned char));

    for (int i = 0; i < rows * columns; i++) {
        importance[i] = abs(edgesX[i]) + abs(edgesY[i]);
    }
}

int findMinOfThree(int a, int b, int c) {
    int minValue = a;

    if (b < minValue) {
        minValue = b;
    }

    if (c < minValue) {
        minValue = c;
    }

    return minValue;
}

unsigned char findMin(unsigned char *energy, int rows, int columns, int i, int j) {
    unsigned char min = 255;
    unsigned char a = 255;
    unsigned char b = energy[(i + 1) * columns + j];
    unsigned char c = 255;
    if (j == 0) {
        c = energy[(i + 1) * columns + j + 1];
    } else if (j == columns - 1) {
        a = energy[(i + 1) * columns + j - 1];
    } else {
        a = energy[(i + 1) * columns + j - 1];
        c = energy[(i + 1) * columns + j + 1];
    }
    if (min > findMinOfThree(a, b, c))
        min = findMinOfThree(a, b, c);
    return min;
}

int findIndex(unsigned char *M, int rows, int columns, int i, int k) {
    unsigned char min = 255;
    unsigned char a = 255;
    unsigned char b = M[(i) * columns + k];
    unsigned char c = 255;
    if (k == 0) {
        c = M[(i) * columns + k + 1];
    } else if (k == columns - 1) {
        a = M[(i) * columns + k - 1];
    } else {
        a = M[(i) * columns + k - 1];
        c = M[(i) * columns + k + 1];
    }
    if (min > findMinOfThree(a, b, c))
        min = findMinOfThree(a, b, c);
    for (int h = k - 1; h <= k + 1; h++)
        if (min == M[i * columns + h])
            return h;
  return -1;
}

void findOptSeam(unsigned char *energy, int rows, int columns, unsigned char *optSeamMask) {
    unsigned char *M = (unsigned char *)malloc(rows * columns * sizeof(unsigned char));
    optSeamMask = (unsigned char *)malloc(rows * columns * sizeof(unsigned char));
    memset(optSeamMask, 0, rows * columns * sizeof(unsigned char));

    // Least pixel-importance to bottom
    for (int j = 0; j < columns; j++)
        M[(rows - 1) * columns + j] = energy[(rows - 1) * columns + j];

    for (int i = rows - 2; i >= 0; i--)
        for (int j = columns - 1; j >= 0; j--) {
            M[i * columns + j] = energy[i * columns + j] + findMin(M, rows, columns, i, j);
        }

    //
    unsigned char min = M[0];
    for (int j = 0; j < columns; j++) {
        if (min > M[j])
            min = M[j];
    }
    int k = -1;
    for (int j = 0; j < columns; j++) {
        if (min == M[j])
            k = j;
    }
    optSeamMask[k] = 1;
    for (int i = 1; i < rows; i++) {
        k = findIndex(M, rows, columns, i, k);
        optSeamMask[i * columns + k] = 1;
    }

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < columns; ++j) {
            printf("%3d ", M[i * columns + j]);
        }
        printf("\n");
    }

    free(M);
}

void removeSeam(uchar3 *energyMatrix, unsigned char *seamMask, int rows, int columns) {
    // Copy giá trị từ ma trận cũ sang ma trận mới, bỏ qua các cột được đánh dấu trong seamMask
    for (int i = 0; i < rows; ++i) {
        int newColIndex = 0;
        for (int j = 0; j < columns; ++j) {
            if (seamMask[i * columns + j] == 0) {
                energyMatrix[i * (columns - 1) + newColIndex] = energyMatrix[i * columns + j];
                ++newColIndex;
            }
        }
    }
}



void seamCarvingImage(uchar3 *energyMatrix, uchar3 *newEnergyMatrix, int rows, int columns, int n)
{
  for(int k=0; k<n; k++)
  {
    
    unsigned char *grayPixels, *edgesX, *edgesY, *importance, *optSeamMaskMatrix;
    // Chuyển đổi từ ảnh RGB sang grayscale

    rgbToGray(energyMatrix, rows, columns, grayPixels);
    convolveX(grayPixels, rows, columns, edgesX);
    // Convolve with y-Sobel filter
    convolveY(grayPixels, rows, columns, edgesY);
    calculateImportance(edgesX, edgesY, rows, columns, importance);

    // Tạo mảng để lưu kết quả
    
    
    // Gọi hàm findOptSeam
    findOptSeam(importance, rows, columns, optSeamMaskMatrix);
    removeSeam(energyMatrix, optSeamMaskMatrix, rows, columns);

    free(grayPixels);
    free(edgesX);
    free(edgesY);
    free(importance);
    free(optSeamMaskMatrix);

    columns--;
  }
  newEnergyMatrix = (uchar3 *)malloc((rows) * (columns-n) * sizeof(uchar3));
  for(int i=0; i<rows;i++)
    for(int j=0; j<columns-n; j++)
      newEnergyMatrix[i*columns+j]=energyMatrix[i*columns+j];
  return ;
}



void blurImg(uchar3 * inPixels, int rows, int columns, float * filter, int filterrows, 
        uchar3 * outPixels,
        bool useDevice=false, dim3 blockSize=dim3(1, 1), int kernelType=1)
{
	if (useDevice == false)
	{
		// for (int r = 0; r < columns; r++)
    //     {
    //         for (int c = 0; c < rows; c++)
    //         {
    //             int i = r * rows + c;
    //             outPixels[i] = 0.299f*inPixels[3 * i] + 0.587f*inPixels[3 * i + 1] + 0.114f*inPixels[3 * i + 2];
    //         }
    //     }
	}
	// else // Use device
	// {
	// 	GpuTimer timer;
		
	// 	printf("\nKernel %i, ", kernelType);
	// 	// Allocate device memories
	// 	uchar3 * d_inPixels, * d_outPixels;
	// 	float * d_filter;
	// 	size_t pixelsSize = rows * columns * sizeof(uchar3);
	// 	size_t filterSize = filterrows * filterrows * sizeof(float);
	// 	CHECK(cudaMalloc(&d_inPixels, pixelsSize));
	// 	CHECK(cudaMalloc(&d_outPixels, pixelsSize));
	// 	if (kernelType == 1 || kernelType == 2)
	// 	{
	// 		CHECK(cudaMalloc(&d_filter, filterSize));
	// 	}

	// 	// Copy data to device memories
	// 	CHECK(cudaMemcpy(d_inPixels, inPixels, pixelsSize, cudaMemcpyHostToDevice));
	// 	if (kernelType == 1 || kernelType == 2)
	// 	{
	// 		CHECK(cudaMemcpy(d_filter, filter, filterSize, cudaMemcpyHostToDevice));
	// 	}
	// 	else
	// 	{
	// 		// TODO: copy data from "filter" (on host) to "dc_filter" (on CMEM of device)

	// 	}

	// 	// Call kernel
	// 	dim3 gridSize((rows-1)/blockSize.x + 1, (columns-1)/blockSize.y + 1);
	// 	printf("block size %ix%i, grid size %ix%i\n", blockSize.x, blockSize.y, gridSize.x, gridSize.y);
	// 	timer.Start();
	// 	if (kernelType == 1)
	// 	{
	// 		// TODO: call blurImgKernel1

	// 	}
	// 	else if (kernelType == 2)
	// 	{
	// 		// TODO: call blurImgKernel2

	// 	}
	// 	else
	// 	{
	// 		// TODO: call blurImgKernel3

	// 	}
	// 	timer.Stop();
	// 	float time = timer.Elapsed();
	// 	printf("Kernel time: %f ms\n", time);
	// 	cudaDeviceSynchronize();
	// 	CHECK(cudaGetLastError());

	// 	// Copy result from device memory
	// 	CHECK(cudaMemcpy(outPixels, d_outPixels, pixelsSize, cudaMemcpyDeviceToHost));

	// 	// Free device memories
	// 	CHECK(cudaFree(d_inPixels));
	// 	CHECK(cudaFree(d_outPixels));
	// 	if (kernelType == 1 || kernelType == 2)
	// 	{
	// 		CHECK(cudaFree(d_filter));
	// 	}
	// }
	
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

void printError(uchar3 * deviceResult, uchar3 * hostResult, int rows, int columns)
{
	float err = computeError(deviceResult, hostResult, rows * columns);
	printf("Error: %f\n", err);
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);
    printf("****************************\n");

}

int main(int argc, char ** argv)
{
	if (argc !=3 && argc != 5)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	printDeviceInfo();

	// Read input image file
	int rows, columns;
	uchar3 * inPixels, *outPixels;
	readPnm(argv[1], rows, columns, inPixels);
	printf("\nImage size (rows x columns): %i x %i\n", rows, columns);

  seamCarvingImage(inPixels, outPixels, rows, columns, 100);
  char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
  writePnm(outPixels, rows, columns-100, concatStr(outFileNameBase, "_host.pnm"));

  // unsigned char *grayPixels, *edgesX,*edgesY;
  // unsigned char *importance;
  // // Chuyển đổi từ ảnh RGB sang grayscale
  // rgbToGray(inPixels, rows, columns, grayPixels);

  // // Write results to files
  //   char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
  //   // Ghi ảnh grayscale vào file

  // convolveX(grayPixels, rows, columns, edgesX);
  // // Convolve with y-Sobel filter
  //   convolveY(grayPixels, rows, columns, edgesY);
  //   calculateImportance(edgesX, edgesY, rows, columns, importance);
  // writePnmGray(concatStr(outFileNameBase, "X_host.pnm"), rows, columns, edgesX);
  // writePnmGray(concatStr(outFileNameBase, "Y_host.pnm"), rows, columns, edgesY);
  // writePnmGray(concatStr(outFileNameBase, "importance_host.pnm"), rows, columns, (unsigned char *)importance);




// 	// Set up a simple filter with blurring effect 
// 	int filterrows = FILTER_rows;
// 	float * filter = (float *)malloc(filterrows * filterrows * sizeof(float));
// 	for (int filterR = 0; filterR < filterrows; filterR++)
// 	{
// 		for (int filterC = 0; filterC < filterrows; filterC++)
// 		{
// 			filter[filterR * filterrows + filterC] = 1. / (filterrows * filterrows);
// 		}
// 	}

// 	// Blur input image not using device
// 	uchar3 * correctOutPixels = (uchar3 *)malloc(rows * columns * sizeof(uchar3)); 
// 	blurImg(inPixels, rows, columns, filter, filterrows, correctOutPixels);
	
//     // Blur input image using device, kernel 1
//     dim3 blockSize(16, 16); // Default
// 	if (argc == 5)
// 	{
// 		blockSize.x = atoi(argv[3]);
// 		blockSize.y = atoi(argv[4]);
// 	}	
// // 	uchar3 * outPixels1 = (uchar3 *)malloc(rows * columns * sizeof(uchar3));
// // 	blurImg(inPixels, rows, columns, filter, filterrows, outPixels1, true, blockSize, 1);
// // 	printError(outPixels1, correctOutPixels, rows, columns);
	
// // 	// Blur input image using device, kernel 2
// // 	uchar3 * outPixels2 = (uchar3 *)malloc(rows * columns * sizeof(uchar3));
// // 	blurImg(inPixels, rows, columns, filter, filterrows, outPixels2, true, blockSize, 2);
// // 	printError(outPixels2, correctOutPixels, rows, columns);
// // ``
// // 	// Blur input image using device, kernel 3
// // 	uchar3 * outPixels3 = (uchar3 *)malloc(rows * columns * sizeof(uchar3));
// // 	blurImg(inPixels, rows, columns, filter, filterrows, outPixels3, true, blockSize, 3);
// // 	printError(outPixels3, correctOutPixels, rows, columns);

//     // Write results to files
//     char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
// 	writePnm(correctOutPixels, rows, columns, concatStr(outFileNameBase, "_host.pnm"));
// 	// writePnm(outPixels1, rows, columns, concatStr(outFileNameBase, "_device1.pnm"));
// 	// writePnm(outPixels2, rows, columns, concatStr(outFileNameBase, "_device2.pnm"));
// 	// writePnm(outPixels3, rows, columns, concatStr(outFileNameBase, "_device3.pnm"));

// 	// Free memories
// 	free(inPixels);
// 	free(filter);
// 	free(correctOutPixels);
// 	// free(outPixels1);
// 	// free(outPixels2);
// 	// free(outPixels3);
}

